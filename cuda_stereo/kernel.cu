#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <opencv2/opencv.hpp>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

cv::Mat disparity_opencv(const cv::Mat &left_image, const cv::Mat &right_image) {
    cv::Mat disparity(left_image.rows, left_image.cols, CV_16U);
    cv::Ptr<cv::StereoBM> stereo_bm =
        cv::StereoBM::create(0 /* num_disparities */, 21 /* block_size */);
    stereo_bm->compute(left_image, right_image, disparity);

    cv::Mat disparity_8bit(left_image.rows, left_image.cols, CV_8U);
    disparity.convertTo(disparity_8bit, CV_8U, 0.2, 0.0);
    return disparity_8bit;
}

float match_block_cpu(int num_pixels,
    const cv::Mat &left_image, int left_x, int left_y,
    const cv::Mat &right_image, int right_x, int right_y) {

    const uint8_t *left_ptr = left_image.ptr(left_y) + left_x;
    const uint8_t *right_ptr = right_image.ptr(right_y) + right_x;

    // Use the SSD cost for now
    // TODO: try out the normalized cross-correlation
    float cost = 0.0f;
    for (int x = 0; x < num_pixels; x++) {
        float diff = (float)left_ptr[x] - right_ptr[x];
        cost += diff * diff;
    }

    return cost;
}

cv::Mat disparity_cpu(const cv::Mat &left_image, const cv::Mat &right_image) {
    cv::Mat output(left_image.rows, left_image.cols, CV_8U);

    const int HALF_BLOCK_SIZE = 10;

    for (int y = 0; y < left_image.rows; y++) {
        // Ignore the left/right borders for now
        for (int x = HALF_BLOCK_SIZE; x < left_image.cols; x++) {
            // Find best matching block in right image
            auto best_d = std::make_tuple(0, std::numeric_limits<float>::infinity());

            for (int potential_d = 0; potential_d < 32; potential_d++) {
                if (x + potential_d + HALF_BLOCK_SIZE >= left_image.cols) {
                    break;
                }
                int block_start = x - HALF_BLOCK_SIZE;
                float cost = match_block_cpu(HALF_BLOCK_SIZE * 2 + 1,
                    left_image, block_start, y,
                    right_image, block_start + potential_d, y);
                cost += potential_d * 0.1f; // Bias toward smaller disparity

                if (cost < std::get<1>(best_d)) {
                    best_d = std::make_tuple(potential_d, cost);
                }
            }

            output.at<uint8_t>(y, x) = std::get<0>(best_d);
        }
    }

    cv::Mat scaled;
    output.convertTo(scaled, -1, 6.0, 0.0);
    return scaled;
}

int main()
{
    /*const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    // Try out OpenCV
    //cv::Mat left_image = cv::imread("tsukuba/scene1.row3.col1.png", cv::IMREAD_GRAYSCALE);
    cv::Mat left_image = cv::imread("easy_stereo/left.png", cv::IMREAD_GRAYSCALE);
    if (left_image.empty()) {
        throw std::runtime_error("Couldn't load image (left_image)");
    }

    //cv::Mat right_image = cv::imread("tsukuba/scene1.row3.col5.png", cv::IMREAD_GRAYSCALE);
    cv::Mat right_image = cv::imread("easy_stereo/right.png", cv::IMREAD_GRAYSCALE);
    if (right_image.empty()) {
        throw std::runtime_error("Couldn't load image (right_image)");
    }

    /*cv::imshow("Left image", left_image);
    cv::waitKey(0);
    cv::imshow("Right image", right_image);
    cv::waitKey(0);*/

    // Compute stereo disparity
    cv::Mat disparity =
        disparity_cpu(left_image, right_image);
    cv::imshow("Disparity", disparity);
    cv::waitKey(0);

    cv::destroyAllWindows();

    return 0;
}